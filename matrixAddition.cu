#include <iostream>
#include <math.h>
#include <hip/hip_runtime.h>


using namespace std;

//kernel that adds two matrices together
__global__ void addMatrices(int* A, int* B, int* C, int n) {

	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < n && col < n) {
		 
		C[row * n + col] = A[row * n + col] + B[row * n + col];
	}

}

// method that fills in matrices A & B with values before performing addition in the kernel
void init_matrices(int* A, int* B, int n) {

	for (int i = 0; i < n; i++) {
		for (int j = 0; j < n; j++) {
			A[i * n + j] = rand() % 20;
			B[i * n + j] = rand() % 10;
		}
	}
}

//printing out the resulting (C) matrix
void print_results(int* A, int* B, int* C, int n) {

	for (int row = 0; row < n; row++) {
		for (int col = 0; col < n; col++) {

			/*
			cout << A[row * n + col] << " " << endl;
			cout << B[row * n + col] << " " << endl;*/
			cout << C[row * n + col] << " ";
		}
		cout << ""<<endl;
	}

}

int main() {

	int n = 10;
	int block_dim = 16;
	size_t bytes = n * n * sizeof(int);
	int* A, * B, * C;


	hipMallocManaged(&A, bytes);
	hipMallocManaged(&B, bytes);
	hipMallocManaged(&C, bytes);

	init_matrices(A, B, n);
    
	dim3 block_size(block_dim, block_dim);

	dim3 grid_size((n + block_dim - 1) / block_dim, (n + block_dim - 1) / block_dim);

	addMatrices<<<grid_size, block_size>>>(A, B, C, n);
	hipDeviceSynchronize();

	hipError_t err = hipGetLastError();

	if (err != hipSuccess) {
		printf("CUDA error encountered: %s\n", hipGetErrorString(err));
		return -1;
	}

	print_results(A, B, C, n);

	hipFree(A);
	hipFree(B);
	hipFree(C);

 return 0;
}



