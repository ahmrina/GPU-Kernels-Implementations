#include <iostream>
#include <hip/hip_runtime.h>

#include <math.h>

using namespace std;

__global__ void vectorAdd(int* a, int* b, int* c, int n) {

	int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (tid < n) {
		c[tid] = a[tid] + b[tid];
	}

}
void print_results(int* a, int* b, int* c, int n) {

	for (int i = 0; i < n; i++) {
		cout << "a " << a[i] <<" + " << " b " << b[i] << " = " << c[i] << " = c" << endl;
	}
}

int main() {
	int id = hipGetDevice(&id);
	int n = 10;
	size_t bytes = n * sizeof(int);
	int* a, * b, * c;

	hipMallocManaged(&a, bytes);
	hipMallocManaged(&b, bytes);
	hipMallocManaged(&c, bytes);

	//fill vectors with values before addition
	for (int i = 0; i < n; i++) {
		a[i] = rand() % 10;
		b[i] = rand() % 10;
	}

	int block_size = 256;
	int grid_size = ceil((float) n / block_size);

	vectorAdd <<<grid_size, block_size>>> (a, b, c, n);
	hipDeviceSynchronize();

	print_results(a, b, c, n);
	return 0;
}