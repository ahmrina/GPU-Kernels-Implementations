#include <iostream>
#include <math.h>
#include <stdio.h>
#include <hip/hip_runtime.h>


using namespace std;

__global__ void prefix_sum(int* A, int n, int* output_arr) {
	 
	int tidx = threadIdx.x;
	extern __shared__ int shared_mem[];

	if (tidx < n) {

	shared_mem[tidx] = A[tidx];

	/*
	else {
		shared_mem[tidx] = 0;
	}*/
		//__syncthreads();


		// round = 1,...,log2(n)
		 // step = 2^(round - 1)
		for (int step = 1; step < blockDim.x; step *= 2) {
			int x = 0;

			if (tidx >= step) {
				x = shared_mem[tidx - step];
			}
			__syncthreads();

			if (tidx >= step) {
				shared_mem[tidx] += x;
			}
			__syncthreads();
		}

	}

		// copy elements back to original array
		if (tidx < n) {
			output_arr[tidx] = shared_mem[tidx];
		}
	
}

void init_arr(int* A, int n) {

	for (int i = 0; i < n; i++) {
		A[i] = rand() % 10;
	}
}

void print(int* arr, int n) {

	for (int i = 0; i < n; i++) {

		cout << arr[i] << " ";
	} cout << "" << endl;
	
}

int main() {

	int n = 10;
	size_t bytes = n * sizeof(int);

	int* A;
	int* output_arr;

	hipMallocManaged(&A, bytes);
	hipMallocManaged(&output_arr, bytes);

	int block_size = 256;
	//int grid_size = ceil((float) n / block_size);
	int shared_mem_size = block_size * sizeof(int);

	init_arr(A, n);

	prefix_sum <<<1, block_size, shared_mem_size>>> (A, n, output_arr);
	hipDeviceSynchronize();

	hipError_t e = hipGetLastError();

	if (e != hipSuccess) {

		cout << "kernel failed: " << hipGetErrorString(e) << endl;
	}

	print(A, n);
	print(output_arr, n);

	hipFree(A);

	return 0;
}
