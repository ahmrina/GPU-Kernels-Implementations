#include <iostream>
#include <math.h>
#include <hip/hip_runtime.h>


using namespace std;

__global__ void multiplication(int *A, int *b, int *c, int n) {

	int row = blockIdx.x * blockDim.x + threadIdx.x;
	//	int column = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < n) {

		int sum = 0;
		
		for (int i = 0; i < n; i++) {
			sum += A[row * n + i] * b[i];
		}
		c[row] = sum;
	}

}


// expects a 2D array and a 1D array
void fill_values(int *A, int *b, int n) {

	for (int i = 0; i < n; i++) {
		for (int j = 0; j < n; j++) {

			A[i * n + j] = rand() % 20;
		} 
		b[i] = rand() % 5;
	}
}


void print_results(int *A, int * b, int * c, int n) {

	for (int i = 0; i < n; i++) {
		for (int j = 0; j < n; j++) {
			cout << A[i * n + j] << " ";
		}
		cout << endl;
	}

	cout << endl;

	for (int i = 0; i < n; i++) {
		cout << b[i] << endl;
	}

	cout << endl;

	for (int i = 0; i < n; i++) {
		cout << c[i]<< endl;
	}


}

int main() {

	int n = 10;
	size_t bytes = n * n * sizeof(int);

	int* A, * b, * c;

	hipMallocManaged(&A, bytes);
	hipMallocManaged(&b, bytes);
	hipMallocManaged(&c, bytes);


	fill_values(A, b, n);

	int block_size = 256;
	int grid_size = ceil((float)n/ block_size);

	multiplication <<<grid_size, block_size>>> (A, b, c, n);

	hipDeviceSynchronize();

	hipError_t e = hipGetLastError();

	if (e != hipSuccess) {
		cout << "kernel failed: " << hipGetErrorString(e) << endl;
		return -1;
	}

	print_results(A, b, c, n);

	
	hipFree(A);
	hipFree(b);
	hipFree(c);
	
	return 0;
}